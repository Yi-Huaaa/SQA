#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <hipblas.h>
#include <mma.h>
#include <stdbool.h>
using namespace nvcuda;
#define IDX2C(i,j,ld) (((j)*(ld))+(i)) 
#define EDGE 32
#define N (EDGE*EDGE) // N = EDGE * EDGE
#define M 16 // 先從16開始
#define TIMES 1//10
#define STEP 100 //100

#define NQuarter N/4
#define MHalf M/2
#define totalBlkNum N/64
#define totalNumFlipOneTime (NQuarter*MHalf)
#define blkNumOnEgde EDGE/8

// Error check macros
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert (hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
int n64Idx (int n){
    int row = n % EDGE; // n 在原graph上的 row idx
    int col = n / EDGE; // n 在原graph上的 col idx
    int n64Idx = row % 8 + (col % 8) * 8; // (col%8) 在64*64大小的Col，(row % 8) 在64*64大小的row
    return n64Idx;
}

int countBlkNum (int a){
    int aRow = a % EDGE;
    int aCol = a / EDGE;
    int aBlkRow = aRow / 8;
    int aBlkCol = aCol / 8;
    int aBlkNum = aBlkRow + aBlkCol * (EDGE / 8); 
    return aBlkNum;   
}

int judgeColor (int a){
    int aRow = a % EDGE; // n 在原graph上的 row idx
    int aCol = a / EDGE; // n 在原graph上的 col idx
    // if (aRow % 2 == 0 && aCol % 2 == 0) { //綠色
    //     return 0;
    // } else if (aRow % 2 != 0 && aCol % 2 == 0) { // 紅色
    //     return 1;
    // } else if (aRow % 2 == 0 && aCol % 2 != 0) { // 藍色
    //     return 2;
    // } else { // 黑色
    //     return 3;
    // }   
    // combine
    return ((aRow%2)+2*(aCol%2));
}
int couplingIdx (int a) {
    int a64Idx = n64Idx(a);
    int aRow = a % EDGE, aCol = a / EDGE; 
    int colorMinus = ((aRow%2)+(aCol%2)*8);// Green: 0, REd: 1, Blue: 8, Black: 9
    int new_a = ((a64Idx - colorMinus) % 8) / 2 + 4 * ((a64Idx - colorMinus) / 16); // new_a: 在64&64的相同顏色中他是第幾個，總共16個 for one color
    int aBlkNum = countBlkNum(a);
    new_a += aBlkNum*16;//block累積起來的相同顏色
    new_a += judgeColor(a)*(N/4);//因為顏色累積起來的相同顏色
    return new_a;
}
int couplingMatrixIdx (int a, int b){
    // 讀檔案的時候就是column major
    // int a64Idx = n64Idx(a);
    // int b64Idx = n64Idx(b);
    // 假設a是col，以a為主，先換成 64-idx
    int aOnCouplings = couplingIdx(a);
    int bOnCouplings = couplingIdx(b);

    int newCouplingPosition = bOnCouplings + aOnCouplings * N; //在64*64裡面的位置
    return newCouplingPosition;
}

int resNum = 0;
void construct_couplings (int a, int b, int w, float *couplings){
    int aBlkNum = countBlkNum(a);
    int bBlkNum = countBlkNum(b);
    int newPosition = 0;
    newPosition = couplingMatrixIdx(a, b);
    couplings[newPosition] = w;
}

int spinMatrixIdx (int n, int m){ //對了
    // 讀檔案的時候就是column major
    int blkNum = countBlkNum(n);
    // printf("n = %d, m = %d, blkNum = %d\n", n, m, blkNum); 對的
    int a64Idx = n64Idx(n);
    // printf("n = %d, m = %d, a64Idx = %d\n", n,m,a64Idx); 對的
    int new_a = 0, spinIdx = 0;
    /*if (aRow % 2 == 0 && aCol % 2 == 0) { //綠色
        printf("Green\n");
        new_a = (a64Idx % 8) / 2 + 4 * (a64Idx / 16);
        if(m % 2 == 0){ // 偶數層
            spinIdx = new_a + 16 * blkNum;//blk累積，color累積
        } else {
            spinIdx = new_a + 16 * blkNum + (16*1)*totalBlkNum*(M/2) * 4;
        }
    } else if (aRow % 2 != 0 && aCol % 2 == 0) { // 紅色
        printf("Red\n");
        new_a = ((a64Idx - 1) % 8) / 2 + 4 * ((a64Idx - 1) / 16);
        if(m % 2 == 0){ // 偶數層
            spinIdx = new_a + 16 * blkNum + (16*1)*totalBlkNum*(M/2) * 1;//blk累積，color累積
        } else {
            spinIdx = new_a + 16 * blkNum + (16*1)*totalBlkNum*(M/2) * 5;
        }
    } else if (aRow % 2 == 0 && aCol % 2 != 0) { // 藍色
        printf("Blue\n");
        new_a = ((a64Idx - 8) % 8) / 2 + 4 * ((a64Idx - 8) / 16);
        if(m % 2 == 0){ // 偶數層
            spinIdx = new_a + 16 * blkNum + (16*1)*totalBlkNum*(M/2) * 2;//blk累積，color累積
        } else {
            spinIdx = new_a + 16 * blkNum + (16*1)*totalBlkNum*(M/2) * 6;
        }
    } else { // 黑色
        printf("Black\n");
        new_a = ((a64Idx - 9) % 8) / 2 + 4 * ((a64Idx - 9) / 16);
        if(m % 2 == 0){ // 偶數層
            spinIdx = new_a + 16 * blkNum + (16*1)*totalBlkNum*(M/2) * 3;//blk累積，color累積
        } else {
            spinIdx = new_a + 16 * blkNum + (16*1)*totalBlkNum*(M/2) * 7;
        }
    }*/
    //合併上述
    int aRow = n % EDGE, aCol = n / EDGE; 
    int colorMinus = ((aRow%2)+(aCol%2)*8);
    new_a = ((a64Idx - colorMinus) % 8) / 2 + 4 * ((a64Idx - colorMinus) / 16);//new_a
    // printf("n = %d, m = %d, new_a = %d\n", n, m, new_a);

    int color = (aRow%2) + 2*(aCol%2); // 0: Green, 1: Red, 2: Blue, 3: Black;
    spinIdx = ((m%2)*4 + color)*(totalNumFlipOneTime);//先訂位大block中: 奇數偶數，再加上color的累積
    // printf("spinIdx, accum big blk, (m/4)*4 + color) = %d,  spinIdx= %d\n", ((m%2)*4 + color),spinIdx);

    spinIdx += blkNum*16*MHalf;//在定位在block中的哪個位置: 先累積block
    spinIdx += (m/2)*16;//，再累積trotters
    spinIdx += new_a; //最後再加上在最裡面最小條時，他排序第幾
    // printf("spinIdxm, accum blkNum, blkNum*16*MHalf = %d, (m/2)*16 = %d, new_a = %d, spinIdx = %d\n",  blkNum*16*MHalf, (m/2)*16, new_a, spinIdx);

    return spinIdx;

}

void check_couplings(float *couplings, float *couplings_fp32){
    // cudaErrCheck (cudaMemcpy(couplings, couplings_fp32, N*N*sizeof(float), cudaMemcpyDeviceToHost));
    printf("\ncheck_couplings:\n");
    // for (int n = 0; n < N; n++){
    //     for(int k = 0; k < N; k++){
    //         printf("%d ", (int)couplings[IDX2C(n,k,N)]);
    //         // if(couplings[IDX2C(n,k,N)] != 0){
    //         //     printf("row = %d, col = %d\n", n, k);
    //         // }
    //     }
    //     printf("\n");
    // }
    for(int blkNum = 0; blkNum < totalBlkNum; blkNum++){
        printf("\nblock  = %d\n", blkNum);
        for (int row = 0; row <256; row++){
            for(int col = 0; col < 256; col++){
                // if(couplings[IDX2C(row,col,N)] != 0){
                //     // printf("row = %d, col = %d\n", row, col);
                //     printf("* ");
                // } else {
                //     printf("  ");
                // }
                int colAdd = blkNum / 4;
                int rowAdd = blkNum % 4;
                printf("%d ", (int)couplings[IDX2C(row+rowAdd*256,col+colAdd*256,N)]);
            }
            printf("\n");
        }
    }
} 

void check_matrixA (float *matrixA){
    printf("matrixA, block 0, innerBlock = 0\n");
    for(int outBlkNum = 0; outBlkNum < totalBlkNum; outBlkNum++){
        printf("out block = %d\n", outBlkNum);
        for(int innerBlkNum = 0; innerBlkNum < totalBlkNum; innerBlkNum++){
            printf("inner block = %d\n", innerBlkNum);
            for(int i = 0; i < 16; i++){
                for(int j = 0; j < 16; j++){
                    printf("%d ", (int)matrixA[IDX2C(i, j, 16) + outBlkNum*256*16 + innerBlkNum*256]);
                }
                printf("\n");
            }      
        }
    }
  
}

void construct_matrixA (float *couplings, float *matrixA){
    // int cnt = 0;
    int count = 0;
    for(int blkNum = 0; blkNum < totalBlkNum; blkNum++){
        // printf("\nblock  = %d\n", blkNum);
        for(int innerBlkNum = 0; innerBlkNum < totalBlkNum; innerBlkNum++){
            // printf("inner block = %d\n", innerBlkNum);
            // cnt = 0;
            for (int row = 0; row <16; row++){
                for(int col = 0; col < 16; col++){
                    int colAdd = blkNum / 4;
                    int rowAdd = blkNum % 4;
                    // printf("%d ",(int)couplings[IDX2C(row+rowAdd*256+innerBlkNum*16, col+colAdd*256+innerBlkNum*16, N)]);
                    // if(couplings[IDX2C(row+rowAdd*256+innerBlkNum*16, col+colAdd*256+innerBlkNum*16, N)] != 0){
                    //     cnt++;
                    // }
                    matrixA[count] = couplings[IDX2C(row+rowAdd*256+innerBlkNum*16, col+colAdd*256+innerBlkNum*16, N)];
                    count ++;
                }
                // printf("\n");
            }  
            // printf("There are %d number != 0\n", cnt);          
        }
    }
    // printf("count = %d\n", count);

    // check_matrixA (matrixA);
}

void construct_spin(float *spin, float *spin_fp32, int total_spins){
    float x;
    for (int n = 0; n < N; n++){
        for(int m = 0; m < M; m++){
            x = ((float)rand()/(float)(RAND_MAX)) * 1.0;    
            spin[IDX2C(n,m,N)] = ((x>=0.5) ? (float)1. : (float)-1.);
        }
    }
}

void check_spin (float *spin, float *spin_fp32){
    cudaErrCheck (hipMemcpy(spin, spin_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost));
    printf("\ncheck_spin:\n");
    for (int n = 0; n < N; n++){
        for(int m = 0; m < M; m++){
            printf("%d ", (int)spin[IDX2C(n,m,N)] );
        }
        printf("\n");
    }
}

void construct_matrixB(float *spin, float *matrixB){
    int matrixBIdx = 0, spinIdx = 0;
    if(M > 16){
        for(int i = 0; i < M*N; i++){
            matrixB[i] = spin[i];
        }
    } else {
        for(int B = 0; B < 8; B++){
            for(int blkNum = 0; blkNum < totalBlkNum; blkNum++){
                for(int i = 0; i < MHalf*16; i++){
                    matrixB[matrixBIdx] = spin[spinIdx];
                    matrixBIdx++;
                    spinIdx++;
                }
                matrixBIdx += (16-MHalf)*16;
                // printf("matrixBIdx = %d\n", matrixBIdx);
            }
        }
    }
}

void check_matrixB(float *matrixB, int trottersMatrixB){
    for(int B = 0; B < 8; B++){
        for(int blkNum = 0; blkNum < totalBlkNum; blkNum ++){
            for(int i = 0; i < 16; i++){
                for(int j = 0; j < 16; j++){
                    printf("%d ", (int)matrixB[j+i*16]);
                }
                printf("\n");
            }
        }
    }
}

void check_delta_H (float *delta_H, float *delta_H_fp32){
    cudaErrCheck ( hipMemcpy(delta_H, delta_H_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost));
    printf("\ncheck print delta_H\n");
    for (int n = 0; n < N; n++){
        for (int m = 0; m < M; m++){
            printf("%d ", (int)delta_H[IDX2C(n,m,N)]);
        }
        printf("\n");
    }
}

int bMatrixIdx (int oriIdx) {
    int newIdx = 0;
    int sameBlkdiffTrotters = 16*MHalf;
    int a = oriIdx / sameBlkdiffTrotters;
    int b = oriIdx % sameBlkdiffTrotters;
    newIdx = a*2*sameBlkdiffTrotters+b;
    return newIdx;
}

void construct_delta_H (hipblasHandle_t cublasHandle, float *matrixA, float *matrixA_fp32, float *matrixB, float *matrixB_fp32, float *delta_H, float *delta_H_fp32) {
    // 有bug
    float alpha = 1.0f, beta = 1.0f;    
    int matrixAIdx = 0; // OKOK
    int matrixBIdx = 0;
    int delta_HIdx = 0;
    int color = 0;
    // even trotters = 0, odd trotters = 1
    for(int evenOdd = 0; evenOdd < 2; evenOdd ++){
        matrixAIdx = 0; // 對
        for(int outBlkNum = 0; outBlkNum < totalBlkNum; outBlkNum++){
            color = outBlkNum/4;
            delta_HIdx = spinMatrixIdx((EDGE*(color/2)+color%2), evenOdd); // 要換顏色，不用換顏色
            matrixBIdx = bMatrixIdx(delta_HIdx); 
            // printf("n_idx = %d, color = %d, delta_HIdx = %d, matrixBIdx = %d\n", (EDGE*(color/2)+color%2), color, delta_HIdx, matrixBIdx);
            for(int innerBlkNum = 0; innerBlkNum < totalBlkNum; innerBlkNum++){
                matrixAIdx += 256;
                matrixBIdx += 256;
                delta_HIdx += MHalf*16;
                cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                        16, 16, 16,
                                        &alpha, 
                                        matrixA_fp32 + matrixAIdx, HIP_R_32F, 16,
                                        matrixB_fp32, HIP_R_32F,  16, 
                                        &beta, 
                                        delta_H_fp32 + delta_HIdx, HIP_R_32F, 16,
                                        HIPBLAS_COMPUTE_32F_PEDANTIC, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
            }
        }
    }
    // printf("final delta_HIdx = %d\n", delta_HIdx);
    cudaErrCheck(hipMemcpy(delta_H, delta_H_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost));
}

void update_delta_H (hipblasHandle_t cublasHandle, float *couplings, float *couplings_fp32, float *spin, float *spin_fp32, float *delta_H, float *delta_H_fp32){
    float alpha = 1.0f, beta = 0.0f;
    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                            N, M, N,
                            &alpha, 
                            couplings_fp32, HIP_R_32F, N,
                            spin_fp32, HIP_R_32F, N, 
                            &beta, 
                            delta_H_fp32, HIP_R_32F, N,
                            HIPBLAS_COMPUTE_32F_PEDANTIC, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

void flip (int timeIdx, float *couplings, float *couplings_fp32, float *spin, float *spin_fp32, float *matrixB, float *matrixB_fp32, float *delta_H, float *delta_H_fp32, float J_perp, float beta) {
    //even
    float delta = 0., zero = 0., twice_spin = 0.;
    int fIdx = timeIdx*totalNumFlipOneTime;
    for(int blk = 0; blk < totalBlkNum; blk ++){
        for(int m = 0; m < M; m+=2){
            for(int n = 0; n < 16; n++){
                gpuErrchk(hipMemcpy(&delta, delta_H_fp32+fIdx, 1*sizeof(float), hipMemcpyDeviceToHost));
                
                int upperIdx = ((m == 0) ? (fIdx + 4*totalNumFlipOneTime + (MHalf-1)*16) : (fIdx + 4*totalNumFlipOneTime-16));
                int lowerIdx = fIdx + 4*totalNumFlipOneTime;

                delta = 2*M*spin[fIdx]*(delta - M*J_perp*(spin[upperIdx] + spin[lowerIdx]));
                // printf("fIdx = %d, bMatrixIdx(%d) = %d\n", fIdx, fIdx, bMatrixIdx(fIdx));

                if ( (-log(rand() / (float) RAND_MAX) / beta) > delta ) {
                    spin[fIdx] = -spin[fIdx];
                    twice_spin = spin[fIdx]; 
                    gpuErrchk(hipMemcpy(matrixB_fp32 + bMatrixIdx(fIdx), &twice_spin, 1*sizeof(float), hipMemcpyHostToDevice));                
                }
                fIdx ++;
            }
        }
    }
    //odd
    delta = 0., zero = 0., twice_spin = 0.;
    fIdx = (timeIdx+4)*totalNumFlipOneTime;
    fIdx = ((timeIdx == 3) ? (4*totalNumFlipOneTime) : ((timeIdx+4)*totalNumFlipOneTime));
    for(int blk = 0; blk < totalBlkNum; blk ++){
        for(int m = 1; m < M; m += 2){
            for(int n = 0; n < 16; n++){
                gpuErrchk(hipMemcpy(&delta, delta_H_fp32+fIdx, 1*sizeof(float), hipMemcpyDeviceToHost));

                int upperIdx = fIdx - 4*totalNumFlipOneTime;
                int lowerIdx = ((m == M-1) ? (fIdx - 4*totalNumFlipOneTime - (MHalf-1)*16) : (fIdx - 4*totalNumFlipOneTime + 16));
     
                delta = 2*M*spin[fIdx]*(delta - M*J_perp*(spin[upperIdx] + spin[lowerIdx]));
                if ( (-log(rand() / (float) RAND_MAX) / beta) > delta ) {
                    spin[fIdx] = -spin[fIdx];
                    gpuErrchk(hipMemcpy(matrixB_fp32 + bMatrixIdx(fIdx), &twice_spin, 1*sizeof(float), hipMemcpyHostToDevice));                
                }
                fIdx ++;
            }
        }
    }
}

float calculate_E (float *couplings, float *couplings_fp32, float *spin, float *spin_fp32){
    // cudaErrCheck(cudaMemcpy(spin, spin_fp32, M*N*sizeof(float), cudaMemcpyDeviceToHost));
    int E = 0;
    for (int i = 0; i < N; i++){
        for (int j = i+1; j < N; j++){
            // E += -spin[IDX2C(i,0,N)]*spin[IDX2C(j,0,N)]*couplings[IDX2C(i,j,N)];
            E += -spin[spinMatrixIdx(i, 0)]*spin[spinMatrixIdx(j, 0)]*couplings[couplingMatrixIdx(i, j)];
        }
    }
    return E;
}

int main (int argc, char *argv[]) {
    hipblasHandle_t cublasHandle;
    hipEvent_t startcublas;
    hipEvent_t stopcublas;

    cudaErrCheck(hipEventCreate(&startcublas));
    cudaErrCheck(hipEventCreate(&stopcublas));
    cublasErrCheck(hipblasCreate(&cublasHandle));
    
    // Initialize couplings
    float *couplings;  
    couplings = (float*)malloc(N*N*sizeof(float));
    memset(couplings, 0, N*N*sizeof(float));

    float *couplings_fp32;
    cudaErrCheck(hipMalloc((void**)&couplings_fp32, N*N*sizeof(float)));

    int couplingResNum = 2*((2*(EDGE-1)*(2*(EDGE-1)+1)) - (2*105)*(N/64));
    // printf("couplingResNum = %d\n", couplingResNum);

    // Read files
    FILE *instance = fopen(argv[1], "r");
    assert(instance != NULL);
    int a, b, w, total_spins, total_couplings;
    fscanf(instance, "%d%d", &total_spins, &total_couplings);
    while (total_couplings --) {
        fscanf(instance, "%d%d%d", &a, &b, &w);
        construct_couplings(a, b, w, couplings);    
        construct_couplings(b, a, w, couplings);    
    }
    fclose(instance);


    // copy couplings to target device
    //這行感覺之後可以槓掉
    cudaErrCheck ( hipMemcpy(couplings_fp32, couplings, N*N*sizeof(float), hipMemcpyHostToDevice));
    // check couplings, OKOK!
    // check_couplings(couplings, couplings_fp32);
    
    float *matrixA;  
    matrixA = (float*)malloc(64*N*sizeof(float));
    memset(matrixA, 0, 64*N*sizeof(float));

    float *matrixA_fp32;
    cudaErrCheck(hipMalloc((void**)&matrixA_fp32, 64*N*sizeof(float)));
    
    construct_matrixA(couplings, matrixA);
    //check matirxA, OKOK
    // check_matrixA (matrixA);
    cudaErrCheck ( hipMemcpy(matrixA_fp32, matrixA, 64*N*sizeof(float), hipMemcpyHostToDevice));

    // Initialize spin
    float *spin;
    spin = (float*)malloc(M*N*sizeof(float));
    memset(spin, 0, M*N*sizeof(float)); // must initialize, since there are some places not 0
    
    float *spin_fp32;
    cudaErrCheck(hipMalloc((void**)&spin_fp32, M*N*sizeof(float)) );
    cudaErrCheck(hipMemcpy(spin_fp32, spin, M*N*sizeof(float), hipMemcpyHostToDevice));

    int trottersMatrixB = ((M > 16) ? (M): (32));
    float *matrixB;
    matrixB = (float*)malloc(trottersMatrixB*N*sizeof(float));
    memset(matrixB, 0, trottersMatrixB*N*sizeof(float)); // must initialize, since there are some places not 0
    
    float *matrixB_fp32;
    cudaErrCheck(hipMalloc((void**)&matrixB_fp32, trottersMatrixB*N*sizeof(float)));
    cudaErrCheck(hipMemcpy(matrixB_fp32, matrixB, trottersMatrixB*N*sizeof(float), hipMemcpyHostToDevice));

    float *delta_H;
    delta_H = (float*)malloc(M*N*sizeof(float));
    memset(delta_H, 0, M*N*sizeof(float));
    
    float *delta_H_fp32;
    cudaErrCheck(hipMalloc((void**)&delta_H_fp32, M*N*sizeof(float)));
    cudaErrCheck(hipMemcpy(delta_H_fp32, delta_H, M*N*sizeof(float), hipMemcpyHostToDevice));

    // TC, using tensor core
    cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH)); 

    
    // goal: test countBlkNum, OKOK!
    // int blkNum = 0;
    // for(int n = 0; n < N; n++){
    //     blkNum = countBlkNum(n);
    //     printf("original = %d, blk = %d\n",n, blkNum);
    // }

    // // goal: test spinMatrixIdx, OKOK!
    construct_spin(spin, spin_fp32, total_spins);
    cudaErrCheck(hipMemcpy(spin_fp32, spin, M*N*sizeof(float), hipMemcpyHostToDevice));
    // check_spin(spin, spin_fp32); 

    
    /*
    goal: test spinMatrix, OKOK!
    int original = 0, spinIdx;
    printf("print trotters' head location:\n");
    for(int m = 0; m < 4; m++){
        original = 0 + m * N;
        spinIdx = spinMatrixIdx(0, m);
        printf("spin = 0, trotter = %d, original = %d, new = %d\n", m, original, spinIdx);
        printf("---\n");

        original = 1 + m * N;
        spinIdx = spinMatrixIdx(1, m);
        printf("spin = 1, trotter = %d, original = %d, new = %d\n", m, original, spinIdx);
        printf("---\n");

        original = 32 + m * N;
        spinIdx = spinMatrixIdx(32, m);
        printf("spin = 32, trotter = %d, original = %d, new = %d\n", m, original, spinIdx);
        printf("---\n");

        original = 33 + m * N;
        spinIdx = spinMatrixIdx(33, m);
        printf("spin = 33, trotter = %d, original = %d, new = %d\n", m, original, spinIdx);
        printf("---\n");  
        printf("@@@@\n");  
    }
    printf("check accum blocknum\n");
    original = 31 + 3 * N;//應該是紅色
    spinIdx = spinMatrixIdx(31, 3);
    printf("should be Red, spin = 31, trotter = %d, original = %d, new = %d\n", 0, original, spinIdx);
    printf("---\n");  
    */


    // Parameters init
    float results[TIMES] = {0.};
    float used_time[TIMES] = {0.};
    float increase = (8 - 1/(float)16) / (float)STEP;
    float G0 = 8.;

    for (int t = 0; t < TIMES; t++) {
        float beta = 1/(float)16; //bete = 1/Time
        
        //init spin
        construct_spin(spin, spin_fp32, total_spins);
        // check spin, OKOK!
        // check_spin(spin, spin_fp32); 
        cudaErrCheck (hipMemcpy(spin_fp32, spin, M*N*sizeof(float), hipMemcpyHostToDevice));

        //construct matrixB
        construct_matrixB(spin, matrixB);
        //check matrixB
        // check_matrixB(matrixB, trottersMatrixB);
        cudaErrCheck(hipMemcpy(matrixB_fp32, matrixB, trottersMatrixB*N*sizeof(float), hipMemcpyHostToDevice));

        // // Construct the initial energy
        construct_delta_H(cublasHandle, matrixA, matrixA_fp32, matrixB, matrixB_fp32, delta_H, delta_H_fp32);
        // // check delta_H
        // check_delta_H(delta_H, delta_H_fp32); 
   
        float initE = calculate_E(couplings, couplings_fp32, spin, spin_fp32);
        printf("time = %d, initE = %f\n", t, initE);

        // Current cost time
        clock_t begin, end;
        begin = clock();

        for (int p = 0; p < STEP; p++) {
            float Gamma = G0*(1.-(float)p/(float)STEP);
            float J_perp = -0.5*log(tanh((Gamma/M)*beta))/beta;
            for(int f = 0; f < 4; f++){ //f: flip
                flip(f, couplings, couplings_fp32, spin, spin_fp32, matrixB, matrixB_fp32, delta_H, delta_H_fp32, J_perp, beta);
                construct_delta_H(cublasHandle, matrixA, matrixA_fp32, matrixB, matrixB_fp32, delta_H, delta_H_fp32);
            }
            float tmpE = calculate_E(couplings, couplings_fp32, spin, spin_fp32);
            printf("step: %d, Energy: %10lf\n", p, tmpE);
         beta += increase;
        } 
        hipDeviceSynchronize();
        
        
        end = clock();
        double duration = (double)(end-begin) / CLOCKS_PER_SEC;

        used_time[t] = duration;

        float E = calculate_E(couplings, couplings_fp32, spin, spin_fp32);
        results[t] = E;
    }   
    

    printf("Final: \n");
    for (int t = 0; t < TIMES; t++){
        printf("TIME: %d,  used time (s): %10lf,  Energy: %10lf\n", t, used_time[t], results[t]);
    }
    float tot_result_time = 0., tot_energy = 0.;
    for(int i = 0; i < TIMES; i++){
        tot_result_time += used_time[i];
        tot_energy += results[i];
    }
    printf("\nAvg time  : %f\n", tot_result_time/TIMES);
    printf("Avg energy: %f\n", tot_energy/TIMES);
 
    hipblasDestroy(cublasHandle);   
    free(couplings);
    free(spin);
    free(delta_H);
    hipFree(couplings_fp32);
    hipFree(spin_fp32);
    hipFree(delta_H_fp32);
    
    return 0;
}