#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <hipblas.h>
#include <mma.h>
#include <omp.h>
#include <stdbool.h>
#include <sys/time.h>
using namespace nvcuda;
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

// SQA parameters
#define N 32768
#define M 128
#define M_2 128

#define TIMES 1
#define STEP 100

// Must be multiples of 16
#define MATRIX_M N
#define MATRIX_K M_2
#define MATRIX_N M

// Error check macros
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert (hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
void usage ();
void check_spin(float *spin);
void check_couplings(float *couplings);
void check_delta_H (float *couplings, float *spin, float *delta_H, float *delta_H_fp32);
void check_matrix_B (float *matrix_B, float *matrix_B_fp32);

void construct_spin(float *spin, float *spin_fp32,int total_spins){
    float x;
    for (int n = 0; n < N; n++){
        for(int m = 0; m < M; m++){
            x = ((float)rand()/(float)(RAND_MAX)) * 1.0;    
            spin[IDX2C(n,m,N)] = ((x>=0.5) ? (float)1. : (float)-1.);
        }
    }
    cudaErrCheck (hipMemcpy(spin_fp32, spin, M*N*sizeof(float), hipMemcpyHostToDevice));
}

void construct_rand_val(float *rand_val, float *rand_val_fp32){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < M; j++){
            rand_val[IDX2C(i,j,N)] = ((float)rand()/(float)(RAND_MAX)) * 1.0;
        }
    }
    cudaErrCheck (hipMemcpy(rand_val_fp32, rand_val, M*N*sizeof(float), hipMemcpyHostToDevice));
}

__global__ void construct_delta_H(half *couplings_fp16, float *spin_fp32, half *delta_H_fp32){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    delta_H_fp32[idx] = 0;
    for (int m = 0; m < M; m++)
        for (int i = 0; i < N; i++)
            delta_H_fp32[IDX2C(idx,m,N)] += (half)((float)couplings_fp16[IDX2C(i,idx,N)]*spin_fp32[IDX2C(i,m,N)]);
}

void update_delta_H(hipblasHandle_t cublasHandle, half *couplings_fp16, half *matrix_B_fp16, half *delta_H_fp16, int which_spin){
    half alpha = 1.0f, beta = 1.0f;    
    unsigned long long int blk_num = which_spin / M_2;
    int loop_iter = (N/32768 == 0) ? 1 : N/32768; 
    int matrix_m = (N > 32768) ? 32768 : N;

    for (int i = 0; i < loop_iter; i++) {
	unsigned long long int coup_idx = blk_num * (N * M_2) + i*32768*M_2;
        cublasErrCheck(hipblasHgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    matrix_m, MATRIX_N, MATRIX_K,
                                    &alpha,
                                    couplings_fp16 + coup_idx, matrix_m,
                                    matrix_B_fp16, MATRIX_K,
                                    &beta,
                                    delta_H_fp16, matrix_m));
    }
}

void construct_lograndval(float *log_rand_val, float *log_rand_val_fp32, hipStream_t stream){
	#pragma omp parallel for num_threads(16)
    for(int i = 0; i < N; i++){
        log_rand_val[IDX2C(i,0,N)] = (-log(((float)rand()/(float)(RAND_MAX)) * 1.0));
    }
	#pragma omp parallel for num_threads(16)
    for (int m = M-1; m >= 1; m--)
        memcpy(&log_rand_val[m*N], &log_rand_val[(m-1)*N], N*sizeof(float));
    cudaErrCheck (hipMemcpyAsync(log_rand_val_fp32, log_rand_val, M*N*sizeof(float), hipMemcpyHostToDevice, stream));
}

float calculate_E (float *spin, float *spin_fp32, half *couplings){
    cudaErrCheck(hipMemcpy(spin, spin_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost));
    float E = 0;
    for (int i = 0; i < N; i++){
        for (int j = i+1; j < N; j++){
            E += -spin[IDX2C(i,0,N)]*spin[IDX2C(j,0,N)]*(float)couplings[IDX2C(i,j,N)];
        }
    }
    return E;
}

__global__ void judge_flipping_com (half *couplings_fp16, half *delta_H_fp16, float *spin_fp32, half *matrix_B_fp16, float *log_rand_val_fp32, int J_perp, float beta, int start_spin){
    int m = blockIdx.x;
    int idx, mb_idx, upper, lower;
    float delta;
    int first_rd_idx = m&1; //even:0, odd:1
    
    extern __shared__ half deltas[];
    deltas[threadIdx.x] = delta_H_fp16[IDX2C(start_spin+threadIdx.x, m, N)];
    
    upper = (m-1) & (M-1);
    lower = (m+1) & (M-1);
        
    // even: 0~M_2/2-1; odd: M_2/2~M_2-1
    #pragma unroll
    for (int n = 0; n < M_2; n++) {
        int nn = start_spin + ((first_rd_idx*(M_2/2) + n)&(M_2-1));
        idx = IDX2C(nn,m,N);
        mb_idx = IDX2C(nn&(M_2-1),m,M_2);            
        delta = deltas[nn&(M_2-1)];
        delta = beta*spin_fp32[idx]*(delta - J_perp*(spin_fp32[IDX2C(nn,upper,N)] + spin_fp32[IDX2C(nn,lower,N)]));
        
        matrix_B_fp16[mb_idx] = 0;
        if ( (log_rand_val_fp32[idx]) > delta ) {
            spin_fp32[idx] = -spin_fp32[idx];
            matrix_B_fp16[mb_idx] = 2*spin_fp32[idx];
            int ii = start_spin + threadIdx.x;
            deltas[threadIdx.x] += (half)((float)couplings_fp16[IDX2C(ii,nn,N)]*(float)matrix_B_fp16[mb_idx]); 
        } 
        __syncthreads();
    }
}

int main(int argc, char* argv[]) {
    if (argc != 2) 
        usage();
    
    //Initialize TC, for check
    hipblasHandle_t cublasHandle;
    hipEvent_t startcublas;
    hipEvent_t stopcublas;

    cudaErrCheck(hipEventCreate(&startcublas));
    cudaErrCheck(hipEventCreate(&stopcublas));
    cublasErrCheck(hipblasCreate(&cublasHandle));
    
    // Initialize couplings
    half *couplings; // cpu    
    couplings = (half*)malloc(N * N * sizeof(half));
    memset(couplings, 0, N*N*sizeof(half));
    
    half *couplings_fp16; 
    cudaErrCheck(hipMalloc((void**)&couplings_fp16, N*N*sizeof(half)));
    
    // Read files
    FILE *instance = fopen(argv[1], "r");
    assert(instance != NULL);
    int a, b, total_spins, total_couplings;
    float w;
    fscanf(instance, "%d%d", &total_spins, &total_couplings);
    while (total_couplings --) {
        fscanf(instance, "%d%d%f", &a, &b, &w);
        //a--;
        //b--;
        couplings[IDX2C(a,b,N)] = w;
        couplings[IDX2C(b,a,N)] = w;
    }
    fclose(instance);

    // copy couplings to target device
    cudaErrCheck ( hipMemcpy(couplings_fp16, couplings, N*N*sizeof(half), hipMemcpyHostToDevice) );
    
    // Initialize spin
    float *spin;
    spin = (float*)malloc(M*N*sizeof(float));
    memset(spin, 0, M*N*sizeof(float)); // must initialize, since there are some places not 0
    
    float *spin_fp32;
    cudaErrCheck ( hipMalloc((void**)&spin_fp32, M*N*sizeof(float)) );
    cudaErrCheck(hipMemcpy(spin_fp32, spin, M*N*sizeof(float), hipMemcpyHostToDevice));

    half *delta_H;
    delta_H = (half*)malloc(M*N*sizeof(half));
    memset(delta_H, 0, M*N*sizeof(half));
    
    half *delta_H_fp16;
    cudaErrCheck(hipMalloc((void**)&delta_H_fp16, M*N*sizeof(half)));
    cudaErrCheck(hipMemcpy(delta_H_fp16, delta_H, M*N*sizeof(half), hipMemcpyHostToDevice));

    half *matrix_B_fp16;
    cudaErrCheck(hipMalloc((void**)&matrix_B_fp16, M*M_2*sizeof(float)));
    
    float *log_rand_val;
    cudaErrCheck(hipHostMalloc((void**)&log_rand_val, M*N*sizeof(float), hipHostMallocDefault));
    // log_rand_val = (float*)malloc(M*N*sizeof(float));
    
    float *log_rand_val_fp32;
    cudaErrCheck(hipMalloc((void**)&log_rand_val_fp32, M*N*sizeof(float)));
    
    
    // Parameters init
    float results[TIMES] = {0.};
    float used_time[TIMES] = {0.};
    float increase = (16 - 1/(float)16) / (float)STEP;
    float G0 = 8.;

    hipStream_t stream1, stream2;
    cudaErrCheck(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
    cudaErrCheck(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));
    cublasErrCheck(hipblasSetStream(cublasHandle, stream2));
    
    float *best_spin;
    best_spin = (float*)malloc(M*N*sizeof(float));
    memset(best_spin, 0, M*N*sizeof(float)); 
    float best_E = 1e9;

    for (int t = 0; t < TIMES; t++) {
        float beta = 1/(float)16; //bete = 1/Time
        
        //init spin
        construct_spin(spin, spin_fp32, total_spins);
        construct_delta_H<<<N/64, 64>>>(couplings_fp16, spin_fp32, delta_H_fp16);
        hipDeviceSynchronize();
            
        // Current cost time
        struct timeval begin, end;
        gettimeofday(&begin, NULL);

        for (int p = 0; p < STEP; p++) {
            
            float Gamma = G0*(1.-(float)p/(float)STEP);
            float J_perp = -M*0.5*log(tanh((Gamma/M)*beta))/beta;
            
            construct_lograndval(log_rand_val, log_rand_val_fp32, stream1);
            for (int n = 0; n < N; n += M_2) {
                judge_flipping_com <<< M, M_2, M_2*sizeof(float), stream2 >>> (couplings_fp16, delta_H_fp16, 
                    spin_fp32, matrix_B_fp16, log_rand_val_fp32, J_perp, 2*M*beta, n);
                update_delta_H(cublasHandle, couplings_fp16, matrix_B_fp16, delta_H_fp16, n);              
            }
            beta += increase;
            
            //printf("curr: %10lf, energy: %10d\n", curr, E);
            /*float E = calculate_E(spin, spin_fp32, couplings);
	    if (E < best_E) {
	        best_E = E;
                memcpy(best_spin, spin, M*N*sizeof(float));
	    }*/
        } 
        hipDeviceSynchronize();
		gettimeofday(&end, NULL);
		double duration = ((end.tv_sec  - begin.tv_sec) * 1000000u +
                         end.tv_usec - begin.tv_usec) / 1.e6;
            
        used_time[t] = duration;
        
        float best_E = calculate_E(spin, spin_fp32, couplings);
        memcpy(best_spin, spin, M*N*sizeof(float));
        results[t] = best_E;

//	for (int i = 0; i < total_spins; i++)
//	    printf("%d ", (int)best_spin[IDX2C(i,0,N)]);
//	printf("%f\n", best_E);
    }
    
    for (int t = 0; t < TIMES; t++){
        printf("TIME: %d,  used time (s): %10lf,  Energy: %10lf\n", t, used_time[t], results[t]);
    }
    float tot_result_time = 0., tot_energy = 0.;
    for(int i = 0; i < TIMES; i++){
        tot_result_time += used_time[i];
        tot_energy += results[i];
    }
    printf("\nAvg time  : %f\n", tot_result_time/TIMES);
    printf("Avg energy: %f\n", tot_energy/TIMES);

    hipblasDestroy(cublasHandle); 
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    free(couplings);
    free(spin);
    free(delta_H);
    hipHostFree(log_rand_val);
    hipFree(couplings_fp16);
    hipFree(spin_fp32);
    hipFree(delta_H_fp16);
    hipFree(matrix_B_fp16);
    hipFree(log_rand_val_fp32);
    
    return 0;
}

void usage (){
    printf("Usage:\n");
    printf("       ./sqa [spin configuration]\n");
    exit(0);
}

void check_spin(float *spin){
    printf("\ncheck_spin:\n");
    for (int n = 0; n < N; n++){
        for(int m = 0; m < M; m++){
            printf("%d ", (int)spin[IDX2C(n,m,N)] );
        }
        printf("\n");
    }
}

void check_couplings(float *couplings){
    printf("\ncheck_couplings:\n");
    for (int n = 0; n < N; n++){
        for(int k = 0; k < N; k++){
            printf("%d ", (int)couplings[IDX2C(n,k,N)] );
        }
        printf("\n");
    }
}

void check_delta_H (float *couplings, float *spin, float *delta_H, float *delta_H_fp32){
    cudaErrCheck ( hipMemcpy(delta_H, delta_H_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost));
    printf("check..., print delta_H\n");
    for (int n = 0; n < N; n++){
        for (int m = 0; m < M; m++){
            printf("%d ", (int)delta_H[IDX2C(n,m,N)]);
        }
        printf("\n");
    }
}

void check_matrix_B (float *matrix_B, float *matrix_B_fp32){
    cudaErrCheck(hipMemcpy(matrix_B, matrix_B_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost));
    printf("check..., matrix_B:\n");
    for (int n = 0; n < N; n++){
        for (int m = 0; m < M; m++){
            printf("%d ", (int)matrix_B[IDX2C(n,m,N)]);
        }
        printf("\n");
    }
}
