#include "hip/hip_runtime.h"
/*
all on GPU version
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <hipblas.h>
#include <mma.h>
using namespace nvcuda;
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

// SQA parameters
#define N 1024
#define M 16 

#define TIMES 1 //10
#define STEP 100 //100

// Must be multiples of 16
#define MATRIX_M 1024
#define MATRIX_K 1024
#define MATRIX_N 16

// Error check macros
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert (hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//GPU function
//如果是layer = 32, 64, ...可以做平行化，因此才移到GPU上面跑跑看
//目前是沒有平行化的
__global__ void prepare_sigma(float *couplings_fp32, float *spin_fp32, float *delta_H_fp32){
    
    hipblasHandle_t cublasHandle;
    hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);   
    float alpha_tc = 1.0f, beta_tc = 0.0f;
    for (int m = 0; m < M; m++){
        for(int n = 0; n < N; n++){
            hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                        MATRIX_M, MATRIX_N, MATRIX_K, 
                        &alpha_tc,
                        couplings_fp32, HIP_R_32F, MATRIX_M,
                        spin_fp32, HIP_R_32F, MATRIX_K,
                        &beta_tc, 
                        delta_H_fp32, HIP_R_32F, MATRIX_M,
                        HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
        }
    }	
}

__global__ void update_sigmas (float* delta_H_fp32, float* couplings_fp32, int new_spin, int idx){
    delta_H_fp32[idx] = delta_H_fp32[idx] - 2*new_spin*couplings_fp32[idx];
}

// CPU functions
void usage ();
void construct_spin(float *spin, int total_spins);
void construct_delta_H(hipblasHandle_t cublasHandle, float *couplings_fp32, float *spin_fp32, float *delta_H_fp32);
void check_delta_H(float *delta_H);
void update_delta_H(int idx, float new_spin, float * delta_H, float *couplings);

int main(int argc, char* argv[]) {
    if (argc != 2) 
        usage();
    
    //Initialize TC, for check
    hipblasHandle_t cublasHandle;
    hipEvent_t startcublas;
    hipEvent_t stopcublas;

    cudaErrCheck(hipEventCreate(&startcublas));
    cudaErrCheck(hipEventCreate(&stopcublas));
    cublasErrCheck(hipblasCreate(&cublasHandle));
    
    // Initialize couplings
    float *couplings; // cpu    
    couplings = (float*)malloc(N * N * sizeof(float));
    memset(couplings, 0, N*N*sizeof(float));
    
    float *couplings_fp32; // tc-32
    cudaErrCheck(hipMalloc((void**)&couplings_fp32, N * N * sizeof(float)));
    
    // Read files
    FILE *instance = fopen(argv[1], "r");
    assert(instance != NULL);
    int a, b, w, total_spins;
    fscanf(instance, "%d%d", &total_spins, &b);
    while (!feof(instance)) {
        fscanf(instance, "%d%d%d", &a, &b, &w);
        assert(a != b);
        a--;
        b--;
        couplings[IDX2C(a,b,N)] = w;
        couplings[IDX2C(b,a,N)] = w;
    }
    fclose(instance);

    // copy couplings to target device
    cudaErrCheck ( hipMemcpy(couplings_fp32, couplings, N*N*sizeof(float), hipMemcpyHostToDevice) );
    printf("couplings:\n");
    for (int i = 0; i < N; i++){
        for (int k = 0; k < N; k++){
            printf("%d ",(int)couplings[IDX2C(i,k,N)]);
        }
        printf("\n");
    }

    // Initialize spin
    float *spin;
    spin = (float*)malloc(M*N*sizeof(float));
    memset(spin, 0, M*N*sizeof(float)); // must initialize, since there are some places not 0
    float *spin_fp32;
    cudaErrCheck ( hipMalloc((void**)&spin_fp32, M*N*sizeof(float)) );

    float *delta_H;
    delta_H = (float*)malloc(M*N*sizeof(float));
    memset(delta_H, 0, M*N*sizeof(float));
    
    float *delta_H_fp32;
    cudaErrCheck(hipMalloc((void**)&delta_H_fp32, M*N*sizeof(float)) );
    cudaErrCheck (hipMemcpy(delta_H_fp32, delta_H, M*N*sizeof(float), hipMemcpyHostToDevice) );

    // TC, using tensor core
    cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH)); 
    
    // Parameters init
    float results[TIMES] = {0.};
    int delta;
    float increase = (8 - 1/(float)16) / (float)STEP;
    float G0 = 8.;
    
    for (int t = 0; t < TIMES; t++) {
        float beta = 1/(float)16; //bete = 1/Time
        
        //init spin
        printf("construct_spin...\n");
        construct_spin(spin, total_spins);
        cudaErrCheck (hipMemcpy(spin_fp32, spin, M*N*sizeof(float), hipMemcpyHostToDevice));
        
        // Current cost-time
        double curr = 0.;
        
        //init dealta_H
        clock_t begin = clock();
        prepare_sigma <<< 1, 1, 0 >>> (couplings_fp32, spin_fp32, delta_H_fp32);
        //construct_delta_H(cublasHandle,couplings_fp32, spin_fp32, delta_H_fp32);
        //這兩行等等要幹掉
        cudaErrCheck ( hipMemcpy(delta_H, delta_H_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost) );
        check_delta_H(delta_H);
        clock_t end = clock();
        
        double duration = (double)(end-begin) / CLOCKS_PER_SEC;
        curr += duration;
        printf("prepare_delta_H current time: %10lf\n", curr);
        
        
        for (int p = 0; p < STEP; p++) {
            float Gamma = G0*(1.-(float)p/(float)STEP);
            float J_perp = -0.5*log(tanh((Gamma/M)*beta))/beta;
            clock_t begin = clock();
            for (int m = 0; m < M; m++) {
                for (int n = 0; n < N; n++) {
                    int idx = IDX2C(n,m,M);
                    gpuErrchk( hipMemcpy(&delta, delta_H_fp32+idx, 1*sizeof(float), hipMemcpyDeviceToHost));
                    int upper = (m == 0 ? M-1 : m-1);
                    int lower = (m == m-1 ? 0 : m+1);
                    //s: spin
                    delta = 2*M*spin[idx]*(delta - M*J_perp*(spin[IDX2C(n,upper,M)] + spin[IDX2C(n,lower,M)]));
                    if ( (-log(rand() / (float) RAND_MAX) / beta) > delta ) {
                        //update_delta_H(idx, -spin[idx], delta_H, couplings);
                        update_sigmas<<<1, 1, 0>>>(delta_H_fp32, couplings_fp32, spin[idx], idx);   
                        spin[idx] = -spin[idx];
                    }
                }
            }
            beta += increase;
            clock_t end = clock();
            double duration = (double)(end-begin) / CLOCKS_PER_SEC;
            curr += duration;
            
            printf("step: %d, curr: %10lf\n", p, curr);
        }  
            
    }
    
    free(couplings);
    free(spin);
    free(delta_H);
    hipFree(couplings_fp32);
    hipFree(spin_fp32);
    hipFree(delta_H_fp32);
    return 0;
}

void usage (){
    printf("Usage:\n");
    printf("       ./sqa [spin configuration]\n");
    exit(0);
}

void construct_spin(float *spin, int total_spins){
    float x;
    for (int n = 0; n < N; n++){
        for(int m = 0; m < M; m++){
            x = ((float)rand()/(float)(RAND_MAX)) * 1.0;    
            spin[IDX2C(n,m,N)] = ((x>=0.5) ? (float)1. : (float)-1.);
        }
    }
    printf("\nconstruct_spin:\n");
    for (int n = 0; n < N; n++){
        for(int m = 0; m < M; m++){
            printf("%f ", spin[IDX2C(n,m,N)] );
        }
        printf("\n");
    }
}

void construct_delta_H(hipblasHandle_t cublasHandle, float *couplings_fp32, float *spin_fp32, float *delta_H_fp32){
    float alpha_tc = 1.0f, beta_tc = 0.0f;
    for (int m = 0; m < M; m++){
        for(int n = 0; n < N; n++){
            cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                        MATRIX_M, MATRIX_N, MATRIX_K, 
                                        &alpha_tc,
                                        couplings_fp32, HIP_R_32F, MATRIX_M,
                                        spin_fp32, HIP_R_32F, MATRIX_K,
                                        &beta_tc, 
                                        delta_H_fp32, HIP_R_32F, MATRIX_M,
                                        HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
        }
    }  
}

void check_delta_H (float* delta_H){
    printf("\ncheck..., print delta_H\n");

    for (int n = 0; n < N; n++){
        for (int m = 0; m < M; m++){
            printf("%f ", delta_H[IDX2C(n,m,N)]);
        }
        printf("\n");
    }
}

void update_delta_H(int idx, float new_spin,float * delta_H, float *couplings){
    delta_H[idx] = delta_H[idx] - 2*new_spin*couplings[idx];
}

