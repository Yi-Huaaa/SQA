#include "hip/hip_runtime.h"
// new method, CPU version
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <hipblas.h>
#include <mma.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
using namespace nvcuda;
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

// SQA parameters
#define N 8192
#define M 16 

#define TIMES 1//10
#define STEP 100 //100

// Must be multiples of 16
#define MATRIX_M 8192
#define MATRIX_K 8192
#define MATRIX_N 16

// Error check macros
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert (hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void usage (){
    printf("Usage:\n");
    printf("       ./sqa [spin configuration]\n");
    exit(0);
}

void construct_spin(float *spin, int total_spins){
    float x;
    for (int n = 0; n < N; n++){
        for(int m = 0; m < M; m++){
            x = ((float)rand()/(float)(RAND_MAX)) * 1.0;    
            spin[IDX2C(n,m,N)] = ((x>=0.5) ? (float)1. : (float)-1.);
        }
    }
}

void construct_delta_H(hipblasHandle_t cublasHandle, float *couplings_fp32, float *spin_fp32, float *delta_H_fp32){
    float alpha = 1.0f, beta = 0.0f;
    cublasErrCheck(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                MATRIX_M, MATRIX_N, MATRIX_K,
                                &alpha, 
                                couplings_fp32, MATRIX_M,
                                spin_fp32, MATRIX_K,
                                &beta,
                                delta_H_fp32, MATRIX_M));

}


void update_delta_H(hipblasHandle_t cublasHandle, float *couplings_fp32, float *matrix_B_fp32, float *delta_H_fp32, int which_spin, float alpha, float beta){
    int blk_num = which_spin / M;
    int coup_idx = blk_num * (N*M);
    cublasErrCheck(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                MATRIX_M, MATRIX_N, MATRIX_N,
                                &alpha, 
                                couplings_fp32 + coup_idx, MATRIX_M,
                                matrix_B_fp32, MATRIX_N,
                                &beta,
                                delta_H_fp32, MATRIX_M));

}

void check_spin(float *spin){
    printf("\ncheck_spin:\n");
    for (int n = 0; n < N; n++){
        for(int m = 0; m < M; m++){
            printf("%d ", (int)spin[IDX2C(n,m,N)] );
        }
        printf("\n");
    }
}

void check_couplings(float *couplings){
    printf("\ncheck_couplings:\n");
    for (int n = 0; n < N; n++){
        for(int k = 0; k < N; k++){
            printf("%d ", (int)couplings[IDX2C(n,k,N)] );
        }
        printf("\n");
    }
}

void check_delta_H (float *couplings, float *spin, float *delta_H, float *delta_H_fp32){
    cudaErrCheck ( hipMemcpy(delta_H, delta_H_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost) );
    //check_couplings(couplings);
    //check_spin(spin);
    printf("check..., print delta_H\n");
    for (int n = 0; n < N; n++){
        for (int m = 0; m < M; m++){
            printf("%d ", (int)delta_H[IDX2C(n,m,N)]);
        }
        printf("\n");
    }
}

void check_matrix_B (float *matrix_B, float *matrix_B_fp32){
    cudaErrCheck(hipMemcpy(matrix_B, matrix_B_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost));
    printf("check..., matrix_B:\n");
    for (int n = 0; n < N; n++){
        for (int m = 0; m < M; m++){
            printf("%d ", (int)matrix_B[IDX2C(n,m,N)]);
        }
        printf("\n");
    }
}

void clear_matrix_B (float *matrix_B, float *matrix_B_fp32){
    memset(matrix_B, 0, M*M*sizeof(float));
    cudaErrCheck(hipMemcpy(matrix_B_fp32, matrix_B, M*M*sizeof(float), hipMemcpyHostToDevice));
}


int main(int argc, char* argv[]) {
	if (argc != 2) 
		usage();
    
    //Initialize TC, for check
    hipblasHandle_t cublasHandle;
    hipEvent_t startcublas;
    hipEvent_t stopcublas;

    cudaErrCheck(hipEventCreate(&startcublas));
    cudaErrCheck(hipEventCreate(&stopcublas));
    cublasErrCheck(hipblasCreate(&cublasHandle));
	
	// Initialize couplings
    float *couplings; // cpu    
    couplings = (float*)malloc(N * N * sizeof(float));
    memset(couplings, 0, N*N*sizeof(float));
    
    float *couplings_fp32; // tc-32
    cudaErrCheck(hipMalloc((void**)&couplings_fp32, N*N*sizeof(float)));
    
	// Read files
    FILE *instance = fopen(argv[1], "r");
    assert(instance != NULL);
    int a, b, w, total_spins, total_couplings;
    fscanf(instance, "%d%d", &total_spins, &total_couplings);
    while (total_couplings --) {
        fscanf(instance, "%d%d%d", &a, &b, &w);
        //assert(a != b);
        a--;
        b--;
        couplings[IDX2C(a,b,N)] = w;
        couplings[IDX2C(b,a,N)] = w;
    }
    fclose(instance);

    // copy couplings to target device
    cudaErrCheck ( hipMemcpy(couplings_fp32, couplings, N*N*sizeof(float), hipMemcpyHostToDevice) );
    
	// Initialize spin
    float *spin;
    spin = (float*)malloc(M*N*sizeof(float));
    memset(spin, 0, M*N*sizeof(float)); // must initialize, since there are some places not 0
    
    float *spin_fp32;
    cudaErrCheck ( hipMalloc((void**)&spin_fp32, M*N*sizeof(float)) );
    cudaErrCheck(hipMemcpy(spin_fp32, spin, M*N*sizeof(float), hipMemcpyHostToDevice));

    float *delta_H;
    delta_H = (float*)malloc(M*N*sizeof(float));
    memset(delta_H, 0, M*N*sizeof(float));
    
    float *delta_H_fp32;
    cudaErrCheck(hipMalloc((void**)&delta_H_fp32, M*N*sizeof(float)));
    cudaErrCheck(hipMemcpy(delta_H_fp32, delta_H, M*N*sizeof(float), hipMemcpyHostToDevice));

    float *matrix_B;
    matrix_B = (float*)malloc(M*M*sizeof(float));
    memset(matrix_B, 0, M*M*sizeof(float));

    float *matrix_B_fp32;
    cudaErrCheck(hipMalloc((void**)&matrix_B_fp32, M*M*sizeof(float)));
    cudaErrCheck(hipMemcpy(matrix_B_fp32, matrix_B, M*M*sizeof(float), hipMemcpyHostToDevice));

	// TC, using tensor core
    cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH)); 
    
    // Parameters init
    float results[TIMES] = {0.};
    float used_time[TIMES] = {0.};
    float delta = 0.;
    float increase = (8 - 1/(float)16) / (float)STEP;
    float G0 = 8.;
    float zero = 0.;
    float twice_spin = 0.;
    
    srand(time(0));
    for (int t = 0; t < TIMES; t++) {
        float beta = 1/(float)16; //bete = 1/Time
        
        //init spin
        construct_spin(spin, total_spins);
        cudaErrCheck (hipMemcpy(spin_fp32, spin, M*N*sizeof(float), hipMemcpyHostToDevice));

        // Current cost time
        double curr = 0.;
        //小心：一定要放在STPE外面，不然每次新一個STEP他又會再重新乘一次導致有問題，因為我的spin_fp32是沒有更新的會有錯誤
        construct_delta_H(cublasHandle,couplings_fp32, spin_fp32, delta_H_fp32);
        //check_delta_H (couplings, spin, delta_H, delta_H_fp32);

        for (int p = 0; p < STEP; p++) {
            float Gamma = G0*(1.-(float)p/(float)STEP);
            float J_perp = -0.5*log(tanh((Gamma/M)*beta))/beta;
            clock_t begin = clock();

            for (int n = 0; n < N; n++) {
                for (int m = 0; m < M; m++) {
                    int idx = IDX2C(n,m,N);
                    int mb_idx = IDX2C(n%M,m,M);
                    
                    gpuErrchk( hipMemcpy (&delta, delta_H_fp32+idx, 1*sizeof(float), hipMemcpyDeviceToHost));
                    int upper = (m == 0 ? M-1 : m-1);
                    int lower = (m == m-1 ? 0 : m+1);
                    delta = 2*M*spin[idx]*(delta - M*J_perp*(spin[IDX2C(n,upper,N)] + spin[IDX2C(n,lower,N)]));
                    if ( (-log(rand() / (float) RAND_MAX) / beta) > delta ) {
                        spin[idx] = -spin[idx];
                        twice_spin = 2*spin[idx];
                        gpuErrchk(hipMemcpy(matrix_B_fp32+mb_idx, &twice_spin, 1*sizeof(float), hipMemcpyHostToDevice));
                    } else {
                        gpuErrchk(hipMemcpy(matrix_B_fp32+mb_idx, &zero, 1*sizeof(float), hipMemcpyHostToDevice));    
                    }

                }
                update_delta_H(cublasHandle, couplings_fp32, matrix_B_fp32, delta_H_fp32, n, 1.0, 1.0);
                
                clear_matrix_B(matrix_B, matrix_B_fp32);
            }

            beta += increase;
            clock_t end = clock();
            double duration = (double)(end-begin) / CLOCKS_PER_SEC;
            curr += duration;

            //隨機取一層就好
            int E = 0;
            for (int i = 0; i < N; i++){
                for (int j = i+1; j < N; j++){
                    //E += -spin[i*M+0]*spin[j*M+0]*couplings[i*N+j];
                    E += -spin[IDX2C(i,0,N)]*spin[IDX2C(j,0,N)]*couplings[IDX2C(i,j,N)];
                }
            }
            results[t] = E;
            used_time[t] = curr;
            //printf("STEP = %d ,energy = %10lf\n", p, results[t]);
        } 
    }
    
    printf("Final: \n");
    for (int t = 0; t < TIMES; t++){
        printf("TIME: %d,  used time (s): %10lf,  Energy: %10lf\n", t, used_time[t], results[t]);
    }
    
    float tot_result_time = 0., tot_energy = 0.;
    for(int i = 0; i < TIMES; i++){
        tot_result_time += used_time[i];
        tot_energy += results[i];
    }
    printf("\nAvg time  : %f\n", tot_result_time/TIMES);
    printf("Avg energy: %f\n", tot_energy/TIMES);

    hipblasDestroy(cublasHandle);   
    free(couplings);
    free(spin);
    free(delta_H);
    hipFree(couplings_fp32);
    hipFree(spin_fp32);
    hipFree(delta_H_fp32);
    return 0;
}



