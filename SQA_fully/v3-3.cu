/*
all on CPU version
delta_H的定義好像是，對於一spin而言，他要算N*M大小的東東
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <hipblas.h>
#include <mma.h>
using namespace nvcuda;
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

// SQA parameters
#define N 32
#define M 16 

#define TIMES 2 //10
#define STEP 100 //100

// Must be multiples of 16
#define MATRIX_M 32
#define MATRIX_K 32
#define MATRIX_N 16

// Error check macros
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert (hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// CPU functions
void usage ();
void construct_spin(float *spin, int total_spins);
void construct_delta_H(hipblasHandle_t cublasHandle, float *couplings_fp32, float *spin_fp32, float *delta_H_fp32);
void check_delta_H(float *delta_H);
void update_delta_H(int idx, float new_spin, float * delta_H, float *couplings);

int main(int argc, char* argv[]) {
	if (argc != 2) 
		usage();
    
    //Initialize TC, for check
    hipblasHandle_t cublasHandle;
    hipEvent_t startcublas;
    hipEvent_t stopcublas;

    cudaErrCheck(hipEventCreate(&startcublas));
    cudaErrCheck(hipEventCreate(&stopcublas));
    cublasErrCheck(hipblasCreate(&cublasHandle));
	
	// Initialize couplings
    float *couplings; // cpu    
    couplings = (float*)malloc(N * N * sizeof(float));
    memset(couplings, 0, N*N*sizeof(float));
    
    float *couplings_fp32; // tc-32
    cudaErrCheck(hipMalloc((void**)&couplings_fp32, N * N * sizeof(float)));
    
	// Read files
    FILE *instance = fopen(argv[1], "r");
    assert(instance != NULL);
    int a, b, w, total_spins;
    fscanf(instance, "%d%d", &total_spins, &b);
    while (!feof(instance)) {
        fscanf(instance, "%d%d%d", &a, &b, &w);
        assert(a != b);
        a--;
        b--;
        couplings[IDX2C(a,b,N)] = w;
        couplings[IDX2C(b,a,N)] = w;
    }
    fclose(instance);

    // copy couplings to target device
    cudaErrCheck ( hipMemcpy(couplings_fp32, couplings, N*N*sizeof(float), hipMemcpyHostToDevice) );
    
    /*printf("couplings:\n");
	for (int i = 0; i < N; i++){
		for (int k = 0; k < N; k++){
			printf("%d ",(int)couplings[IDX2C(i,k,N)]);
		}
		printf("\n");
	}*/
    
	// Initialize spin
    float *spin;
    spin = (float*)malloc(M*N*sizeof(float));
    memset(spin, 0, M*N*sizeof(float)); // must initialize, since there are some places not 0
    float *spin_fp32;
    cudaErrCheck ( hipMalloc((void**)&spin_fp32, M*N*sizeof(float)) );

    float *delta_H;
    delta_H = (float*)malloc(M*N*sizeof(float));
    memset(delta_H, 0, M*N*sizeof(float));
    
    float *delta_H_fp32;
    cudaErrCheck(hipMalloc((void**)&delta_H_fp32, M*N*sizeof(float)) );
    cudaErrCheck (hipMemcpy(delta_H_fp32, delta_H, M*N*sizeof(float), hipMemcpyHostToDevice) );

	// TC, using tensor core
    cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH)); 
    
    // Parameters init
    float results[TIMES] = {0.};
    float used_time[TIMES] = {0.};
    float delta = 0.;
    float increase = (8 - 1/(float)16) / (float)STEP;
    float G0 = 8.;
    
    
    for (int t = 0; t < TIMES; t++) {
        float beta = 1/(float)16; //bete = 1/Time
        
        //init spin
        //printf("construct_spin...\n");
        construct_spin(spin, total_spins);
        cudaErrCheck (hipMemcpy(spin_fp32, spin, M*N*sizeof(float), hipMemcpyHostToDevice));
        
        // Current cost-time
        double curr = 0.;
        
        //init dealta_H
        clock_t begin = clock();
        construct_delta_H(cublasHandle,couplings_fp32, spin_fp32, delta_H_fp32);
        cudaErrCheck ( hipMemcpy(delta_H, delta_H_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost) );
        check_delta_H(delta_H);
        clock_t end = clock();
        
        double duration = (double)(end-begin) / CLOCKS_PER_SEC;
        curr += duration;
        printf("preapre_delta_H_curr: %10lf\n", curr);
        
        
        for (int p = 0; p < STEP; p++) {
            float Gamma = G0*(1.-(float)p/(float)STEP);
            float J_perp = -0.5*log(tanh((Gamma/M)*beta))/beta;
            clock_t begin = clock();
            for (int m = 0; m < M; m++) {
                for (int n = 0; n < N; n++) {
                    int idx = IDX2C(n,m,M);
                    gpuErrchk( hipMemcpy(&delta, delta_H_fp32+idx, 1*sizeof(float), hipMemcpyDeviceToHost));
                    int upper = (m == 0 ? M-1 : m-1);
                    int lower = (m == m-1 ? 0 : m+1);
                    delta = 2*M*spin[idx]*(delta - M*J_perp*(spin[IDX2C(n,upper,M)] + spin[IDX2C(n,lower,M)]));
                    if ( (-log(rand() / (float) RAND_MAX) / beta) > delta ) {
                        //printf("flipping: m = %d, n = %d, delta = %f\n", m, n, delta);
                        update_delta_H(idx, spin[idx], delta_H, couplings);
                        spin[idx] = -spin[idx];
                    }
                }
            }
            beta += increase;
            clock_t end = clock();
            double duration = (double)(end-begin) / CLOCKS_PER_SEC;
            curr += duration;

            //隨機取一層就好
            int E = 0;
            for (int i = 0; i < N; i++){
                for (int j = i+1; j < N; j++){
                    E += -spin[i*M+0]*spin[j*M+0]*couplings[i*N+j];
                }
            }
            results[t] = E;
            used_time[t] = curr;
            //printf("step: %d, curr: %10lf, energy: %10d\n", p, curr, E);
        }  
    }
    
    printf("Final: \n");
    for (int t = 0; t < TIMES; t++){
        printf("TIME: %d,  used time (s): %10lf, Energy: %10lf\n", t, used_time[t], results[t]);
    }
    
    free(couplings);
    free(spin);
    free(delta_H);
    hipFree(couplings_fp32);
    hipFree(spin_fp32);
    hipFree(delta_H_fp32);
    return 0;
}

void usage (){
    printf("Usage:\n");
    printf("       ./sqa [spin configuration]\n");
    exit(0);
}

void construct_spin(float *spin, int total_spins){
	float x;
    for (int n = 0; n < N; n++){
        for(int m = 0; m < M; m++){
            x = ((float)rand()/(float)(RAND_MAX)) * 1.0;    
            spin[IDX2C(n,m,N)] = ((x>=0.5) ? (float)1. : (float)-1.);
        }
    }
	/*printf("\nconstruct_spin:\n");
    for (int n = 0; n < N; n++){
	    for(int m = 0; m < M; m++){
    		printf("%f ", spin[IDX2C(n,m,N)] );
    	}
    	printf("\n");
    }*/
}

void construct_delta_H(hipblasHandle_t cublasHandle, float *couplings_fp32, float *spin_fp32, float *delta_H_fp32){
    float alpha_tc = 1.0f, beta_tc = 0.0f;
	for (int m = 0; m < M; m++){
		for(int n = 0; n < N; n++){
			cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
			            				MATRIX_M, MATRIX_N, MATRIX_K, 
				         				&alpha_tc,
				          				couplings_fp32, HIP_R_32F, MATRIX_M,
			                			spin_fp32, HIP_R_32F, MATRIX_K,
			                			&beta_tc, 
			               				delta_H_fp32, HIP_R_32F, MATRIX_M,
			               				HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
		}
	}  
}

void check_delta_H (float* delta_H){
	printf("\ncheck..., print delta_H\n");

    for (int n = 0; n < N; n++){
    	for (int m = 0; m < M; m++){
    		printf("%f ", delta_H[IDX2C(n,m,N)]);
    	}
    	printf("\n");
    }
}

void update_delta_H(int idx, float new_spin,float * delta_H, float *couplings){
    delta_H[idx] = delta_H[idx] - 2*new_spin*couplings[idx];
}

